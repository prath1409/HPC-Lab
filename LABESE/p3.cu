#include "hip/hip_runtime.h"
%%writefile p3.cu
#include<stdio.h>
#include<hip/hip_runtime.h>
#define row1 2 
#define col1 3 
#define row2 3 
#define col2 4 

__global__ void matproductsharedmemory(int *l,int *m, int *n)
{
    int x=blockIdx.x;
    int y=blockIdx.y;
    __shared__ int p[col1];

    int i;
    int k=threadIdx.x;

    n[col2*y+x]=0;

   p[k]=l[col1*y+k]*m[col2*k+x];

  __syncthreads();

  for(i=0;i<col1;i++)
  n[col2*y+x]=n[col2*y+x]+p[i];
}

int main()
{
    int a[row1][col1] = {{1,2,3}, {4,5,6}};
    int b[row2][col2] = {{1,2,3,4}, {4,5,2,3}, {8,9,10,11}};
    int c[row1][col2];
    int *d,*e,*f;
    int i,j;

   hipMalloc((void **)&d,row1*col1*sizeof(int));
    hipMalloc((void **)&e,row2*col2*sizeof(int));
    hipMalloc((void **)&f,row1*col2*sizeof(int));

 hipMemcpy(d,a,row1*col1*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(e,b,row2*col2*sizeof(int),hipMemcpyHostToDevice);

dim3 grid(col2,row1);
/* Here we are defining two dimensional Grid(collection of blocks) structure. Syntax is dim3 grid(no. of columns,no. of rows) */

matproductsharedmemory<<<grid,col1>>>(d,e,f);

 hipMemcpy(c,f,row1*col2*sizeof(int),hipMemcpyDeviceToHost);

 printf("\n Product of two matrices:\n ");
    for(i=0;i<row1;i++)
    {
        for(j=0;j<col2;j++)
        {
              printf("%d\t",c[i][j]);
        }
        printf("\n");
    }

    hipFree(d);
    hipFree(e);
    hipFree(f);

    return 0;
}
